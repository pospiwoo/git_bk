#include "hip/hip_runtime.h"
#include <stdio.h>

#define N 2
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

using namespace std;

class vecarray{
    public:
        int *vecptr[N];                //array of pointers pointing to array
        int dim[N];                     //store length of each array pointed to

        __device__ __host__ vecarray(); //constructor
        __device__ __host__ int sum();  //sum up all the elements in the array being
                                       //pointed to
};

vecarray::vecarray(){
    for(int i = 0; i<N; i++)
    {
        vecptr[i] = NULL;
        dim[i] = 0;
    }
}

__device__ __host__ int vecarray::sum(){
    int i=0, j=0, s=0;
    for (i=0; i<N; i++)
        for(j=0; j < dim[i]; j++)
            s += vecptr[i][j];
    return s;
}

__global__ void addvecarray( vecarray * v, int *s){
    *s = v->sum();
}

int main(){                                 //copy *V to device, do sum() and pass back
    vecarray *v, *dev_v;                    //the result by dev_v
    v = new vecarray;
    int a[3] = {1,2,3};                     //initialize v manually
    int b[4] = {4,5,6,7};
    int result = 0;
    int *dev_result;
    v->vecptr[0] = a;
    v->vecptr[1] = b;
    v->dim[0] = 3; v->dim[1] = 4;
    int *vptr[N];

    hipMalloc((void**)&dev_v, sizeof(vecarray));
    cudaCheckErrors("cudaMalloc1 fail");
    hipMemcpy(dev_v, v, sizeof(vecarray),hipMemcpyHostToDevice); //copy class object
    cudaCheckErrors("cudaMemcpy1 fail");

    for(int i = 0; i < N; i++){
        hipMalloc((void**)&(vptr[i]), v->dim[i]*sizeof(int));
        cudaCheckErrors("cudaMalloc2 fail");
        hipMemcpy(&(dev_v->vecptr[i]), &vptr[i], sizeof(int*), hipMemcpyHostToDevice);
        cudaCheckErrors("cudaMemcpy2 fail");
    }

    for(int i = 0; i<N; i++ ){                   //copy arrays
        hipMemcpy(vptr[i], v->vecptr[i], v->dim[i]*sizeof(int), hipMemcpyHostToDevice);
        cudaCheckErrors("cudaMemcpy3 fail");
    }
    hipMalloc((void **)&dev_result, sizeof(int));
    cudaCheckErrors("cudaMalloc3 fail");
    addvecarray<<<1,1>>>(dev_v, dev_result);

    hipMemcpy(&result, dev_result, sizeof(int), hipMemcpyDeviceToHost);
    cudaCheckErrors("cudaMemcpy4 fail");
    printf("the result is %d\n", result);
    return 0;
}

