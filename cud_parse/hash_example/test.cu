#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "hash_map_template.h"
using namespace CUDASTL;

__global__ void foo(HashMap<int,int> * h){
	int i=get_thread_id();
//	int i=threadIdx.x;
	(*h)[i]=i*10;
}

__global__ void bar(HashMap<int,int> * h, int * arr){
//	int i=get_thread_id();
//	arr[i]=(*h)[i];
	for(HashMap<int,int>::Iterator it=h->begin();it!=h->end();++it){
		arr[it->key]=it->value;		
	}

}

const int numBlocks=10;
const int numThreads=10;
const int N=numThreads*numBlocks;

int main(){

	int * d_arr;
	hipMalloc((void **)&d_arr,sizeof(int)*N);
	HashMap<int,int> * h=CreateHashMap<int,int,HashFunc<int> >(79, 2000);

	foo<<<numBlocks,numThreads>>> (h);

	hipDeviceSynchronize();
//	bar<<<numBlocks,numThreads>>>(h,d_arr);
	bar<<<1,1>>>(h,d_arr);
	int * h_arr=(int *)malloc(sizeof(int)*N);
	hipMemcpy(h_arr,d_arr,sizeof(int)*N,hipMemcpyDeviceToHost);

	int i;
	for(i=90;i<100;i++){
		printf("%d\t",h_arr[i]);
		if(i%8==7)
			printf("\n");
	}
	printf("\n");

	return 0;
}

